#include "hip/hip_runtime.h"
/* Ordinary differential equation solver, Runge-Kutta-England technique.
   Copyright © 1988 Free Software Foundation, Inc.
   François Pinard <pinard@iro.umontreal.ca>, 1988.

   This program is free software; you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation; either version 1, or (at your option)
   any later version.

   This program is distributed in the hope that it will be useful, but
   WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
   General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program; if not, write to the Free Software Foundation,
   Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
*/



#include <stdio.h>
#include <math.h>
#include <stdlib.h>

typedef struct struct_rke_variables
{

    /* The following are saved from rke_init call arguments. */

    int n_equations;		/* Number of simultaneous equations */
    int (*eval_routine) ();	/* Routine to compute derivatives */

    /* These may be changed by the user between two solve calls. */

    double minimum_step;	/* Minimum allowable step size */
    double maximum_step;	/* Maximum allowable step size */
    double current_step;	/* Current integration step size */
    double error_slope;		/* Slope of maximum error per time unit */
    double error_biais;		/* Biais of maximum error per time unit */
    int accepted_steps;		/* Accumulated number of accepted steps */
    int rejected_steps;		/* Accumulated number of rejected steps */
}
rke_variables;


extern double fabs ();

/* Initialize a new system of equations. */

void rke_init (int number, int (*routine) () , rke_variables * var)	/* Newly allocated reentrancy block */
{

  (var)->n_equations = number;
  (var)->eval_routine = routine;
  (var)->minimum_step = 0.000001;
  (var)->maximum_step = 1000000.0;
  (var)->current_step = 1.0;
  (var)->error_slope = 0.0000001;
  (var)->error_biais = 0.00000001;
  (var)->accepted_steps = 0;
  (var)->rejected_steps = 0;

  return;
}

/* Terminate a set of equations. */

void rke_term (rke_variables * var)
{
  free (var);
}


/* Main routine of the module, ODE solver. */

/* Perform a consistent move of time in the system. */

int	 rke_solve (rke_variables * var, double *time, double variables[], double aimed_time, double * scrap_mem)
{
  double whole_step;		/* Signed integration step size */
  double quarter_step;		/* 0.25 * whole_step */
  double half_step;		/* 0.50 * whole_step */
  double three_quarter_step;	/* 0.75 * whole_step */
  double estimated_error;	/* Error as estimated by England method */
  double allowable_error;	/* Maximum error that user permits */
  int within_tolerance;		/* Allowable error has not been passed */
  int all_errors_small;		/* All errors within 2% of tolerances */
  int length_of_array;		/* Length of temporary arrays, is bytes */
  int k;			/* Index in various arrays */

  double *dp, *vt, *v, *d;
  double *a1, *a2, *a3, *a4, *a5, *a6, *a7;

  /* Allocate the work arrays. */

  length_of_array = var->n_equations * sizeof (double);

  dp = &scrap_mem[0];
  vt = &scrap_mem[length_of_array];
  v = &scrap_mem[2*length_of_array];
  d = &scrap_mem[3*length_of_array];
  a1 = &scrap_mem[4*length_of_array];
  a2 = &scrap_mem[5*length_of_array];
  a3 = &scrap_mem[6*length_of_array];
  a4 = &scrap_mem[7*length_of_array];
  a5 = &scrap_mem[8*length_of_array];
  a6 = &scrap_mem[9*length_of_array];
  a7 = &scrap_mem[10*length_of_array];

  /* The integration will continue if a minimum step could bring the
     system closer to the time that is aimed for, even if we have to
     overshoot it a little. */

  while (2 * fabs (aimed_time - *time) > var->minimum_step)
  {

      /* Evaluate initial step size and direction. */

    if ((whole_step = aimed_time - *time) > 0.0)
    {
     if (whole_step > var->current_step)
       whole_step = var->current_step;
   }
   else
   {
     if (whole_step < - var->current_step)
       whole_step = - var->current_step;
   }

      /*  Evaluate initial differentials. */

   if (! (*(var->eval_routine)) (*time, variables, dp))
     return 0;

   do

	/* Loop integrating at this time point until integration error is
	   within tolerances.  In any case, adjust integration step size. */

   {
	  /* Calculate various step sizes. */

     quarter_step = 0.25 * whole_step;
     half_step = quarter_step + quarter_step;
     three_quarter_step = half_step + quarter_step;

	  /* Perform a partial computation for one step of Runge-Kutta
	     4th order integration, as far as necessary to chain it to
	     England method for estimating integration errors. */

     for (k = 0; k < var->n_equations; ++k)
     {
       a1[k] = half_step * dp[k];
       v[k] = variables[k]
       + 0.5*a1[k];
     }

     if (! (*(var->eval_routine)) (*time + quarter_step, v, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a2[k] = half_step * d[k];
       v[k] = variables[k]
       + 0.25 * (a1[k] + a2[k]);
     }

     if (! (*(var->eval_routine)) (*time + quarter_step, v, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a3[k] = half_step * d[k];
       v[k] = variables[k]
       + (-a2[k] + a3[k] + a3[k]);
     }

     if (! (*(var->eval_routine)) (*time + half_step, v, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a4[k] = half_step * d[k];
       vt[k] = variables[k]
       + (a1[k] + 4.0*a3[k] + a4[k]) / 6.0;
     }

     if (! (*(var->eval_routine)) (*time + half_step, vt, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a5[k] = half_step * d[k];
       v[k] = vt[k]
       + 0.5*a5[k];
     }

     if (! (*(var->eval_routine)) (*time + three_quarter_step, v, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a6[k] = half_step * d[k];
       v[k] = vt[k]
       + 0.25*(a5[k] + a6[k]);
     }

     if (! (*(var->eval_routine)) (*time + three_quarter_step, v, d))
       return 0;

     for (k = 0; k < var->n_equations; ++k)
     {
       a7[k] = half_step * d[k];
       v[k] = variables[k]
       + (-a1[k] - 96.0*a2[k] + 92.0*a3[k] - 121.0*a4[k]
         + 144.0*a5[k] + 6.0*a6[k] - 12.0*a7[k]) / 6.0;
     }

	  /* Perform England error analysis on partial integration. */

     if (! (*(var->eval_routine)) (*time + whole_step, v, d))
       return 0;

     within_tolerance = 1;
     all_errors_small = 1;

     for (k = 0; k < var->n_equations; ++k)
     {
       estimated_error
       = fabs ((-a1[k] + 4.0*a3[k] + 17.0*a4[k]
         - 23.0*a5[k] + 4.0*a7[k] - half_step*d[k])
       / 90.0);
       allowable_error = fabs (whole_step)
       * (var->error_slope*fabs (vt[k]) + var->error_biais);
       if (estimated_error > allowable_error)
       {
        within_tolerance = 0;
        break;
      }
      else if (estimated_error > 0.02 * allowable_error)
        all_errors_small = 0;
    }
    if (within_tolerance)
    {
	    ++(var->accepted_steps);

	      /* Complete the Runge-Kutta step and return values. */

	    for (k = 0; k < var->n_equations; ++k)
		    v[k] = vt[k] + (-a6[k] + a7[k] + a7[k]);
	    if (! (*(var->eval_routine)) (*time + whole_step, v, d))
		    return 0;
	    *time += whole_step;
	    for (k = 0; k < var->n_equations; ++k)
		    variables[k] = vt[k]
		    + (a5[k] + 4.0*a7[k] + half_step*d[k]) / 6.0;

	      /* Increment step size if desirable. */
	    if (all_errors_small && fabs (whole_step) == var->current_step){
		    if (2 * var->current_step > var->maximum_step){
			    var->current_step = var->maximum_step;
		    }
		    else{
			    var->current_step *= 2;
		    }
	    }
    }
	else
	{
		++var->rejected_steps;
		/* Decrement step size if possible. */
		if (fabs (whole_step) > var->minimum_step)
		{
			if (var->current_step < 2 * var->minimum_step)
				var->current_step = var->minimum_step;
			else
				var->current_step *= 0.5;
			if (aimed_time > *time)
				whole_step = var->current_step;
			else
				whole_step = - var->current_step;
		}
		else
			return 0;	/* Convergence failed */
	}
   }
	  while (!within_tolerance);
	  {
		  return 1;			/* Convergence succeeded */
	  }




/* Check how close we can get back to our initial conditions. */

void print_return (double back,double initial)
{
  printf ("  returning to %12.6lf, got %12.6lf\n", initial, back);
  return;
}



/* Print statistics about number of steps. */

void print_steps(rke_variables * var)
{
  printf ("    using %3d accepted and %3d rejected steps\n",
    var->accepted_steps, var->rejected_steps);
  return;
}


/* Integration under a normal curve. */



static double example_1_const;	/* 1.0 / sqrt (2 * pi) */


static int problem_function_1 (  double t  , double v[1]  , double d[1])
{
  d[0] = example_1_const * exp (-0.5 * t * t);
  return 1;
}


static void example_1 ()
{

  rke_variables* p = malloc(sizeof(struct struct_rke_variables) );
  double * scrap_mem = (double *) malloc(11*sizeof(double));
  
  double t;
  double v[1];

  example_1_const = 1.0 / sqrt (2 * 3.1415926);

  rke_init (1, problem_function_1, p);

  t = -1.0;			/* Start at -1.0 */
  v[0] = 0.0;			/* Surface is 0.0 at this point */

  /* Now, simply move to +1.0, and collect the answer. */

  if (rke_solve (p, &t, v, 1.0 , scrap_mem))
    printf ("\nProbability	= %12.6lf.\n", v[0]);
  else
    printf ("\nProbability not computed, error.\n");
  print_steps (p);

  /* Just undo this, to see if we get back where we started. */

  if (rke_solve (p, &t, v, -1.0 , scrap_mem))
    print_return (v[0], 0.0);
  else
    printf ("  return to start not computed, error.\n");
  print_steps (p);

  rke_term (p);
  free(scrap_mem);
  return;
}



/* Rediscovering cos and sin. */

static int problem_function_2 (double t, double v[2], double d[2])
{
  d[0] = -v[1];
  d[1] = v[0];
  return 1;
}


static void example_2 ()
{
  rke_variables* p = malloc(sizeof(struct struct_rke_variables) );
  double * scrap_mem = (double *) malloc(2*11*sizeof(double));

  double t;
  double v[2];

  rke_init (2, problem_function_2 , p);

  t = 0.0;			/* Start where we know the values */
  v[0] = 1.0;			/* cos 0 = 1.0 */
  v[1] = 0.0;			/* sin 0 = 0.0 */

  /* Now, simply move to 1.5, and collect the answer. */

  if (rke_solve (p, &t, v, 1.5,scrap_mem))
    printf ("\ncos (1.5)	= %12.6lf.\n", v[0]);
  else
    printf ("\ncos (1.5) not computed, error.\n");
  print_steps (p);

  /* Just undo this, to see if we get back where we started. */

  if (rke_solve (p, &t, v, 0.0 , scrap_mem))
  {
    print_return (v[0], 1.0);
    print_return (v[1], 0.0);
  }
  else
    printf ("  return to start not computed, error.\n");
  print_steps (p);

  rke_term (p);
  free(scrap_mem);
  return;
}


/* Box slowing by friction in air. */

static int problem_function_3 (double t, double v[2], double d[2])
{
  d[0] = v[1];
  d[1] = -0.01 * v[1] * v[1];
  return 1;
}


static void example_3 () {
  
  rke_variables* p = malloc(sizeof(struct struct_rke_variables) );
  double * scrap_mem = (double *) malloc(2*11*sizeof(double));

  double t;
  double v[2];

  rke_init (2, problem_function_3, p);

  t = 0.0;			/* Start the clock... */
  v[0] = 0.0;			/* ... with no distance so far */
  v[1] = 100.0;		/* ... but some initial speed */

  /* Now, simply ask the clock to be 5.0, and collect the answer. */

  

  if (rke_solve (p, &t, v, 5.0 , scrap_mem))
    printf ("\nDistance	= %12.6lf.\n", v[0]);
  else
    printf ("\nDistance not computed, error.\n");
  print_steps (p);

  /* Just undo this, to see if we get back where we started. */

  if (rke_solve (p, &t, v, 0.0 , scrap_mem))
  {
    print_return (v[0], 0.0);
    print_return (v[1], 100.0);
  }
  else
    printf ("  return to start not computed, error.\n");
  print_steps (p);

  rke_term (p);
  free(scrap_mem);
  return;
}


/* Main program. */

int main ()
{
  example_1 ();
  example_2 ();
  example_3 ();
  return 0 ;
}
